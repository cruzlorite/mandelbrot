#include "hip/hip_runtime.h"
// Copyright(C) 2020 José María Cruz Lorite
//
// This file is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with this project.  If not, see <https://www.gnu.org/licenses/>.

#include <chrono>
#include <iostream>
#include <vector>

// CUDA hip/hip_vector_types.h from NVIDIA repository on github
// https://github.com/NVIDIA/cuda-samples/blob/master/Common/hip/hip_vector_types.h
#include "hip/hip_vector_types.h"

// export/import png images
#include "util/lodepng.h"

// Handy macro
#define check(func, msg) __check((func), msg, __FILE__, __LINE__)

// Check if something goes wrong
void __check(hipError_t err, const std::string& msg, const std::string& file, int line) {
    if (err != hipSuccess) {
        std::cerr << msg << ". File '" << file << "', line " << line <<std::endl;
        exit(EXIT_FAILURE);
    }
}

// Print properties for particular device
void printDeviceProp(int dev) {
    // get device properties
    hipDeviceProp_t prop;
    check( hipGetDeviceProperties(&prop, dev), "Error getting device properties");

    std::cout << "NVIDIA Corporation, " << prop.name << ", CUDA " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Number of muliprocessors: " << prop.multiProcessorCount << std::endl;
    std::cout << "Warp size in threads:     " << prop.warpSize<< std::endl;
    std::cout << "Max threads per block:    " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max block dimension:      " << prop.maxThreadsDim[0] << " x " <<
                                                 prop.maxThreadsDim[1] << " x " <<
                                                 prop.maxThreadsDim[2] << ""  << std::endl;
    std::cout << "Max grid dimension:       " << prop.maxGridSize[0] << " x " <<
                                                 prop.maxGridSize[1] << " x " <<
                                                 prop.maxGridSize[2] << ""  << std::endl;
}

// CUDA Mandelbrot kernel
__global__
void mandelbrot(
    uint32_t* img,      // Output image
    int width,          // Image width
    int height,         // Image height 
    uint32_t* palette,  // Color palette
    int paletteSize,    // Number of palette colors
    float2 min,         // Complex min value
    float2 delta,       // max - min
    uint maxDepth,      // Max iterations
    float scaleForce)   // Modifies how color palette is used
{
    // Get global invocation id
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Interpolate complex value for this pixel
    float2 c = min + delta * make_float2(
                                x / (float)(width - 1),
                                y / (float)(height - 1));

    int depth = 0;
    float2 z = c;
    float2 z2 = z * z;
    for(; depth < maxDepth && z2.x + z2.y < 4.0f; ++depth) {
        z = make_float2(z2.x - z2.y, 2 * z.x * z.y) + c;
        z2 = z * z;
    }

    int index = y * width + x;
    if (depth == maxDepth) {
        img[index] = 0xFF000000; // Black
    } else {
        // Iteration normalized
        float i = (float)(depth) / (float)(maxDepth - 1);
        float ie = fmin(1.0f, log(i * scaleForce + 1.0f) / log(scaleForce));

        int paletteIndex = floor((1.0f - ie) * (float)(paletteSize - 1));
        img[index] = palette[paletteIndex];
    }
}

int main(void) {
    /******** PARAMETERS *********/
    int  blockSize      = 32;
    int  maxDepth       = 1024;
    int  width          = 1024 * 1;
    int  height         = 1024 * 1;
    float scaleForce    = 20.0f;
    /*****************************/

    // Allocate space for the image on host
    std::vector<uint8_t> img(width * height * 4);

    // Load color palette from file
    unsigned int width_palette = 0;
    unsigned int paletteSize = 0;
    std::vector<uint8_t> palette;
    lodepng::decode(palette, width_palette, paletteSize, "palette.png");

    // Complex plane min max
    float min_real = -2.0;
    float max_real = 1.0f;
    float min_imag = ((max_real - min_real) * height / width) / -2.0f;
    float max_imag = -1.0f * min_imag;

    float2 min      {min_real, min_imag};
    float2 delta    {max_real - min_real, max_imag - min_imag};

    // On device buffers
    uint32_t *img_dev, *palette_dev;

    // Get start time
    auto start = std::chrono::high_resolution_clock::now();

    // Print CUDA properties
    int count; 
    check( hipGetDeviceCount(&count), "Error getting device count");

    for (int i = 0; i < count; ++i) {
        printDeviceProp(i);
        std::cout << std::endl;
    }

    // Allocate memopry on device
    check( hipMalloc(&img_dev, img.size()), "Error allocating memory on device" ); 
    check( hipMalloc(&palette_dev, palette.size()), "Error allocating memory on device" ); 

    // Copy palette to device
    check( hipMemcpy(palette_dev, palette.data(), palette.size(), hipMemcpyHostToDevice),
        "Error copying palette from host to device");

    // Kernel invocation
    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);

    mandelbrot<<<dimGrid, dimBlock>>>(
        img_dev,
        width,
        height,
        palette_dev,
        paletteSize,
        min,
        delta,
        maxDepth,
        scaleForce);

    // Read back from device
    check( hipMemcpy(img.data(), img_dev, img.size(), hipMemcpyDeviceToHost),
        "Error copying image from device to host");

    // Free resources
    hipFree(img_dev);
    hipFree(palette_dev);

    // Get end time
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Time = " <<
        std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000.0f <<
        " seconds" << std::endl;

    // To file
    lodepng::encode("mandelbrot.png", img.data(), width, height);

    exit(EXIT_SUCCESS);
}
